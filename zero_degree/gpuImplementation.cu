#include "hip/hip_runtime.h"
#ifndef CPU_IMPLEMENTATION

#include <cstdio>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_select.cuh>

#include "zeroDegree.hpp"
#include "configuration.hpp"

#define DEBUG_KERNEL
#include "cudaUtil.cuh"

#define THREADS_PER_BLOCK 128

#define INTEGER_DIV_CEIL(A, B) ((A + (B-1)) / B)

__global__ void detectBends(const float* __restrict__ pX, const float* __restrict__ pY, const float* __restrict__ pZ, uint32_t numPoints, uint8_t* __restrict__ bends) {
    __shared__ float shared[(THREADS_PER_BLOCK + REG_MAX_CONV_POINTS) * 3];

    // int dbg_blk = 0;

    float * const s_x = &shared[0];
    float * const s_y = &shared[THREADS_PER_BLOCK + REG_MAX_CONV_POINTS];
    float * const s_z = &shared[(THREADS_PER_BLOCK + REG_MAX_CONV_POINTS) * 2];

    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int beginningOfBlockIdx = (blockIdx.x * blockDim.x);

    int pad = REG_MAX_CONV_POINTS/2;

    // Step 1 - Copy relevant points into shared memory
    // Wrap around when we reach the ends of the point array, rather
    // than filling in with zeroes

    // Fill in front padding with first pad threads
    if(threadIdx.x < pad) {
        int padIdx = beginningOfBlockIdx - pad + (int)threadIdx.x;

        if(padIdx < 0)
            padIdx += numPoints;

        // if(blockIdx.x == dbg_blk)
        //     KERN_DBG("Filling front padding [%d] with index %d\n", idx, padIdx);
        s_x[threadIdx.x] = pX[padIdx];
        s_y[threadIdx.x] = pY[padIdx];
        s_z[threadIdx.x] = pZ[padIdx];
    }
    
    // Fill in middle with all threads
    int fillIdx = idx >= numPoints ? idx - numPoints : idx;
    // if(blockIdx.x == dbg_blk)
    //     KERN_DBG("Filling [%d] with index %d\n", threadIdx.x + pad, fillIdx);
    s_x[threadIdx.x + pad] = pX[fillIdx];
    s_y[threadIdx.x + pad] = pY[fillIdx];
    s_z[threadIdx.x + pad] = pZ[fillIdx];

    // Fill in back padding with last pad threads
    if(threadIdx.x >= blockDim.x - pad) {
        int padIdx = idx + pad;

        if(padIdx >= numPoints)
            padIdx -= numPoints;

        // if(blockIdx.x == dbg_blk)
        //     KERN_DBG("Filling back [%d] with index %d\n", threadIdx.x + 2*pad, padIdx);

        s_x[threadIdx.x + 2*pad] = pX[padIdx];
        s_y[threadIdx.x + 2*pad] = pY[padIdx];
        s_z[threadIdx.x + 2*pad] = pZ[padIdx];
    }

    __syncthreads();

    // Step 2 - Perform r-squared convolution
    float sumXY = 0;
    float sumX = 0;
    float sumY = 0;
    float sumXSquared = 0;
    float sumYSquared = 0;

    float radius = hypotf(s_x[threadIdx.x + pad], s_y[threadIdx.x + pad]);
    int n = (int) (REG_POINTS_PER_INV_METER * (1/radius));

    if(n > REG_MAX_CONV_POINTS)
        n = REG_MAX_CONV_POINTS;

    // TO-DO: Check if this is actually necessary
    if(n % 2 == 0)
        n += 1;

    for(int k = -n/2; k <= n/2; k++) {
        int convI = threadIdx.x + pad + k;

        sumXY += s_x[convI] * s_y[convI];
        sumX += s_x[convI];
        sumY += s_y[convI];
        sumXSquared += s_x[convI] * s_x[convI];
        sumYSquared += s_y[convI] * s_y[convI];
    }

    float r_squared = ((n*sumXY - sumX*sumY)*(n*sumXY - sumX*sumY))
                    / ((n*sumXSquared - (sumX*sumX))
                    * (n*sumYSquared - (sumY*sumY)));

#ifdef PRINT_R_SQUARED
    printf("%f, %f, %f, %f\n", s_x[threadIdx.x + pad], s_y[threadIdx.x + pad], s_z[threadIdx.x + pad], r_squared);
#endif

    // Distance from previous point
    float dist = hypotf(s_x[threadIdx.x + pad] - s_x[threadIdx.x + pad - 1], s_y[threadIdx.x + pad] - s_y[threadIdx.x + pad - 1]);

    // if(r_squared < R_SQUARED_THRESHOLD || dist >= DIST_TOLERANCE) {
    //     KERN_DBG("[%d] Bend Detected (R^2=%f, Dist=%f)\n", idx, r_squared, dist);
    // }
    
    bends[idx] = r_squared < R_SQUARED_THRESHOLD || dist >= DIST_TOLERANCE;
}

__global__ void filterValidSegments(uint32_t* __restrict__ lengths, const uint32_t* __restrict__ offsets, const uint8_t* __restrict__ bends, uint32_t numRuns) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(idx < numRuns && (bends[offsets[idx]] || lengths[idx] < MIN_SEGMENT_LENGTH)) {
        lengths[idx] = 0;
    }
}

// Transforms offset and length data to array-of-structures
__global__ void lengthsAndOffsetsToSegmentDescs(uint32_t* lengths, uint32_t* offsets, segment_desc_t *segmentDescs, uint32_t numInitialSegments) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(idx < numInitialSegments) {
        segmentDescs[idx].segmentStart = offsets[idx];
        segmentDescs[idx].segmentEnd = offsets[idx] + lengths[idx];
    }
}

void runLengthEncodeBends(uint8_t *d_bends, uint32_t *d_offsets, uint32_t *d_lengths, uint32_t *d_numSegments, uint32_t numPoints) {
    size_t cub_temp_storage_req = 0;
    hipcub::DeviceRunLengthEncode::NonTrivialRuns(
            NULL,
            cub_temp_storage_req,
            d_bends,
            d_offsets,
            d_lengths,
            d_numSegments,
            numPoints);

    void *d_cubTempStorage;
    CHECK_CUDA(hipMallocManaged((void **) &d_cubTempStorage, cub_temp_storage_req, hipMemAttachGlobal));

    hipcub::DeviceRunLengthEncode::NonTrivialRuns(
            d_cubTempStorage,
            cub_temp_storage_req,
            d_bends,
            d_offsets,
            d_lengths,
            d_numSegments,
            numPoints);

    CHECK_CUDA(hipFree(d_cubTempStorage));
}

struct NonZeroSegmentLength
{
    HIPCUB_RUNTIME_FUNCTION __device__ __forceinline__
    void NonZeroLength() {}

    HIPCUB_RUNTIME_FUNCTION __device__ __forceinline__
    bool operator()(const segment_desc_t &a) const {
        return a.segmentStart != a.segmentEnd;
    }
};

void condenseSegments(segment_desc_t *segmentDescs, uint32_t *d_numSegments) {
    NonZeroSegmentLength select_op;

    size_t cub_temp_storage_req = 0;
    hipcub::DeviceSelect::If(
            NULL,
            cub_temp_storage_req,
            segmentDescs,
            segmentDescs,
            d_numSegments,
            *d_numSegments,
            select_op);

    void *d_cubTempStorage;
    CHECK_CUDA(hipMalloc((void **) &d_cubTempStorage, cub_temp_storage_req));

    hipcub::DeviceSelect::If(
            d_cubTempStorage,
            cub_temp_storage_req,
            segmentDescs,
            segmentDescs,
            d_numSegments,
            *d_numSegments,
            select_op);

    CHECK_CUDA(hipFree(d_cubTempStorage));
}

uint32_t mergeSegments(segment_desc_t *segmentDescs, uint32_t *d_numSegments) {
    return 0;
}


int planeExtract(float *pX, float *pY, float *pZ, uint32_t numPoints, segment_desc_t **segmentDescs, uint32_t *numSegmentDesc) {
    // Limit bounds of convolution to neighboring blocks
    assert(REG_MAX_CONV_POINTS/2 <= THREADS_PER_BLOCK);

    unsigned int num_blocks = INTEGER_DIV_CEIL(numPoints, THREADS_PER_BLOCK);

    uint8_t *d_bends;
    CHECK_CUDA(hipMallocManaged((void **) &d_bends, sizeof(uint8_t) * numPoints, hipMemAttachGlobal));

    detectBends<<<num_blocks, THREADS_PER_BLOCK>>>(pX, pY, pZ, numPoints, d_bends);
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    uint32_t *d_offsets;
    uint32_t *d_lengths;
    uint32_t *d_numSegments;

    CHECK_CUDA(hipMallocManaged((void **) &d_numSegments, sizeof(uint32_t), hipMemAttachGlobal));

    // Can get away with allocating numPoints/2 since lengths need to be at least 2
    CHECK_CUDA(hipMallocManaged((void **) &d_offsets, sizeof(uint32_t) * numPoints/2, hipMemAttachGlobal));
    CHECK_CUDA(hipMallocManaged((void **) &d_lengths, sizeof(uint32_t) * numPoints/2, hipMemAttachGlobal));

    runLengthEncodeBends(d_bends, d_offsets, d_lengths, d_numSegments, numPoints);

    uint32_t numInitialSegments = *d_numSegments;

    num_blocks = INTEGER_DIV_CEIL(numInitialSegments, THREADS_PER_BLOCK);
    filterValidSegments<<<num_blocks, THREADS_PER_BLOCK>>>(d_lengths, d_offsets, d_bends, numInitialSegments);
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipFree(d_bends));

    CHECK_CUDA(hipMallocManaged((void **) segmentDescs, sizeof(segment_desc_t) * numInitialSegments, hipMemAttachGlobal));

    lengthsAndOffsetsToSegmentDescs<<<num_blocks, THREADS_PER_BLOCK>>>(d_lengths, d_offsets, *segmentDescs, numInitialSegments);
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    // No longer needed, as their information is now in segmentDescs
    CHECK_CUDA(hipFree(d_offsets));
    CHECK_CUDA(hipFree(d_lengths));

    // Condense segments so long as merging reduces the number of segments
    do {
        condenseSegments(*segmentDescs, d_numSegments);
    } while(mergeSegments(*segmentDescs, d_numSegments));

    CHECK_CUDA(hipFree(d_numSegments));

    *numSegmentDesc = *d_numSegments;

    return 0;
}

#endif